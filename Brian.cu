#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <unistd.h>
#include "Brian.v1.h"
#include "Brian.v0.h"
//On=1,Off=0,Dying=2
int SIZE, ITERATIONS, ANIMATE, BLOCKS, THREADS, SEED, UNOPTIMIZED, PRINT, live_cells, dead_cells, dying_cells;
void print_board(int board[], int size, int iteration)
{
	if (iteration != -1)
	{
		printf("Iteration %d\n", iteration);
	}
	for (int i = 0;i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			if (board[i * size + j] == 1)//if it is alive
			{
				printf("\u25A3 ");
				live_cells++;
			}
			else if(board[i * size +j] == 0)//if it is dead
			{
				printf("\u25A2 ");
				dead_cells++;
			}
			else if(board[i * size +j] == 2)//if it is dying
			{
				printf("\u25A7 ");
				dying_cells++;
			}
		}
		printf("\n");
	}
	printf("Live Cells =%d ,Dead Cells =%d Dying Cells =%d\n\n",live_cells, dead_cells, dying_cells);
	live_cells = 0;
	dead_cells = 0;
	dying_cells = 0;
}

void arg_parse(int argc, char *argv[])
{
	int i = 1;
	char c;
	while(i < argc)
	{
		sscanf(argv[i++], "%c", &c);
		if (c == 's')//matrix size
		{
			sscanf(argv[i++], "%d", &SIZE);
		}
		if (c == 'a')//animation or not
		{
			ANIMATE = 1;
			printf("fu");
		}
		if (c == 'i')//iterations
		{
			sscanf(argv[i++], "%d", &ITERATIONS);
		}
		if (c == 'b')//number of blocks
		{
			sscanf(argv[i++], "%d", &BLOCKS);
		}
		if (c == 't')//number of threads
		{
			sscanf(argv[i++], "%d", &THREADS);
		}
		if (c == 'e')//random seed(?)
		{
			sscanf(argv[i++], "%d", &SEED);
		}
		if (c == 'u')//version using global memory
		{
			UNOPTIMIZED = 1;
		}
		if (c == 'p')//print board
		{
			sscanf(argv[i++], "%d", &PRINT);
		}
	}
}

int run()
{
	// run arguments
	int animate = ANIMATE != -1 ? ANIMATE : false; // variable for animation--default False
	int size = SIZE ? SIZE : 64;//matrix size--default 64
	int iterations = ITERATIONS ? ITERATIONS : 6;//generations--default 6
	int no_blocks = BLOCKS ? BLOCKS : size; //number of blocks--default 64
	int no_threads = THREADS ? THREADS : size;//number of thread--default 64
	int unoptimized_run = UNOPTIMIZED ? UNOPTIMIZED : 0;//variable for version--default optimized
	int print = PRINT != -1 ? PRINT : true;

	// Initialize random seed
	srand(SEED != -1 ? SEED : time(NULL));

	// host(cpu) memory
	int *input = (int*)calloc(size * size, sizeof(int));//matrix for production-initialisation
	int *output = (int*)calloc(size * size, sizeof(int));//the matrix we print
	int *devin, *devout, *devtemp;//matrix of gpu

	// device(gpu) memory
	hipMalloc((void**)&devin, size * size * sizeof(int));//matrix for production-initialisation
	hipMalloc((void**)&devout, size * size * sizeof(int));//the matrix we print
	hipMalloc((void**)&devtemp, size * size * sizeof(int));//matrix of next generation

	// production and initialisation of the universe
	for (int i = 0;i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			input[i*size + j] = rand() % 3;// a number from 0,2
		}
	}

	if (print)
		print_board(input, size, 0);

	// initial matrix migration from cpu to gpu
	hipMemcpy(devin, input, size * size * sizeof(int), hipMemcpyHostToDevice);

	//the matrix we print
	hipMemcpy(devout, output, size * size * sizeof(int), hipMemcpyHostToDevice);

	//used when the size of shared memory is unknown during the compile
	//dynamic memory allocation in shared memory
	//it is used only from version 2
	//containts threads data of a block
	int shared_board_size = (no_threads + 2 * size) * sizeof(int);

	// timer start
	struct timeval  tv1, tv2;
	gettimeofday(&tv1, NULL);

	// choose version
	// 1st version with global memmory
	if (unoptimized_run)
	{

		for (int i = 0;i<iterations;i++)
		{
			if (i == 0)
			{
				//start calculations with first production-initialisation matrix
				play<<<no_blocks,no_threads>>>(devin, devout);
			}
			else
			{
				//continue calculations with next generation matrix
				play<<<no_blocks,no_threads>>>(devtemp, devout);
			}
			//migration of next generation matrix to output matrix of gpu(inside og gpu)
			hipMemcpy(devtemp, devout, size * size * sizeof(int), hipMemcpyDeviceToDevice);

			//migration of output matrix from gpu to cpu
			hipMemcpy(output, devout, size * size * sizeof(int), hipMemcpyDeviceToHost);

			//print results
			if (animate == true)
			{
				system("clear");
				print_board(output, size, i);
				usleep(100000);
			}
		}
		printf("Unoptimized run\n");
	}
	//2nd version with shared memmory,uses a 3rd matrix for calculations
	else
	{
		for (int i = 0;i<iterations;i++)
		{
			if (i == 0)
			{
				//start calculations with first production-initialisation matrix
				play_with_shared_memory<<<no_blocks,no_threads,shared_board_size>>>(devin, devout, size);
			}
			else
			{
				//continue calculations with next generation matrix
				play_with_shared_memory<<<no_blocks,no_threads,shared_board_size>>>(devtemp, devout, size);
			}
			//migration of next generation matrix to output matrix of gpu(inside og gpu)
			hipMemcpy(devtemp, devout, size * size * sizeof(int), hipMemcpyDeviceToDevice);
			//migration of output matrix from gpu to cpu
			hipMemcpy(output, devout, size * size * sizeof(int), hipMemcpyDeviceToHost);

			//print results
			if (animate == true)
			{
				system("clear");
				print_board(output, size, i);
				usleep(100000);
			}
		}
	}

	// migration of result from gpu to cpu
	hipMemcpy(output, devout, size * size * sizeof(int), hipMemcpyDeviceToHost);

	if (print)
		print_board(output, size, iterations);

	// calculate the run time
	gettimeofday(&tv2, NULL);
	printf ("Total time in kernel = %f seconds\n",(double) (tv2.tv_usec - tv1.tv_usec) / 1000000 + (double) (tv2.tv_sec - tv1.tv_sec));



	// Free device memory
	hipFree(devin);
	hipFree(devout);
	hipFree(devtemp);


    return 0;
}

int main(int argc, char* argv[])
{
	SIZE = 0, ITERATIONS = 0, ANIMATE = -1, BLOCKS = 0, THREADS = 0, UNOPTIMIZED = 0, SEED = -1, PRINT = -1;
	arg_parse(argc, argv);
	run();
	return 0;
}
