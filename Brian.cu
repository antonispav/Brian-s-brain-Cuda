#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <unistd.h>
#include "Brian.v1.h"
#include "Brian.v0.h"
//On=1,Off=0,Dying=2 
int SIZE, ITERATIONS, ANIMATE, BLOCKS, THREADS, SEED, UNOPTIMIZED, PRINT, live_cells, dead_cells, dying_cells;
void print_board(int board[], int size, int iteration)
{
	if (iteration != -1)
	{
		printf("Iteration %d\n", iteration);
	}
	for (int i = 0;i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			if (board[i * size + j] == 1)//an einai alive
			{
				printf("\u25A3 ");
				live_cells++;
			}
			else if(board[i * size +j] == 0)//an einai dead
			{
				printf("\u25A2 ");
				dead_cells++;
			}
			else if(board[i * size +j] == 2)//an einai dying
			{
				printf("\u25A7 ");
				dying_cells++;
			}
		}
		printf("\n");
	}
	printf("Live Cells =%d ,Dead Cells =%d Dying Cells =%d\n\n",live_cells, dead_cells, dying_cells);
	live_cells = 0;
	dead_cells = 0;
	dying_cells = 0;
}

void arg_parse(int argc, char *argv[])
{
	int i = 1;
	char c;
	while(i < argc)
	{
		sscanf(argv[i++], "%c", &c);
		if (c == 's')//to megethos tou pinaka
		{
			sscanf(argv[i++], "%d", &SIZE);
		}
		if (c == 'a')//animation h oxi
		{
			ANIMATE = 1;
			printf("fu");
		}
		if (c == 'i')//epanalipseis
		{
			sscanf(argv[i++], "%d", &ITERATIONS);
		}
		if (c == 'b')//number of blocks
		{
			sscanf(argv[i++], "%d", &BLOCKS);
		}
		if (c == 't')//number of threads
		{
			sscanf(argv[i++], "%d", &THREADS);
		}
		if (c == 'e')//random seed(?)
		{
			sscanf(argv[i++], "%d", &SEED);
		}
		if (c == 'u')//version using global memory
		{
			UNOPTIMIZED = 1;
		}
		if (c == 'p')//print board
		{
			sscanf(argv[i++], "%d", &PRINT);
		}
	}
}

int run()
{
	// orismata gia to run tou programm
	int animate = ANIMATE != -1 ? ANIMATE : false; // metavliti gia to an tha uparxei h oxi animation
	int size = SIZE ? SIZE : 64;//megethos pinaka--default 64
	int iterations = ITERATIONS ? ITERATIONS : 6;//genies--default 6
	int no_blocks = BLOCKS ? BLOCKS : size; //arithmos twn block--default 64
	int no_threads = THREADS ? THREADS : size;//arithmos twn thread--default 64
	int unoptimized_run = UNOPTIMIZED ? UNOPTIMIZED : 0;//metavliti gia to pia ekdosi epilegthike--default optimized
	int print = PRINT != -1 ? PRINT : true;

	// Initialize random seed
	srand(SEED != -1 ? SEED : time(NULL));

	// desmeush mnhmhs ston host(cpu)
	int *input = (int*)calloc(size * size, sizeof(int));//pinakas gia paragwgh-arxikopoihsh
	int *output = (int*)calloc(size * size, sizeof(int));//o pinakas pou emfanizete
	int *devin, *devout, *devtemp;//pinakes ths gpu

	//desmeush mnhmhs ston device(gpu)
	hipMalloc((void**)&devin, size * size * sizeof(int));//pinakas gia paragwgh-arxikopoihsh
	hipMalloc((void**)&devout, size * size * sizeof(int));//o pinakas pou emfanizete
	hipMalloc((void**)&devtemp, size * size * sizeof(int));//o pinakas epomenhs genias

	// paragwgh kai arxikopoihsh sympantos
	for (int i = 0;i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			input[i*size + j] = rand() % 3;// enas arithmos apo to 0,2
		}
	}

	if (print)
		print_board(input, size, 0);

	// antigrafi tou pinaka arxikopoihshs apo cpu->gpu
	hipMemcpy(devin, input, size * size * sizeof(int), hipMemcpyHostToDevice);

	//o pinakas pou emfanizete
	hipMemcpy(devout, output, size * size * sizeof(int), hipMemcpyHostToDevice);

	//xrhsimopoieitai otan den einai gnwsto to megethos ths shared memory kata thn metaglwtish tou programmatos
	//desmeuei dynamika mnhmh sthn shared memory
	// xrhsimopoieitai mono apo thn 2h ekdosh
	//periexei ta stoixeia twn threads enos block
	int shared_board_size = (no_threads + 2 * size) * sizeof(int);

	// xekina to xronometro
	struct timeval  tv1, tv2;
	gettimeofday(&tv1, NULL);

	// dialexe ekdosh
	// 1h ekdosi me global memmory
	if (unoptimized_run)
	{

		for (int i = 0;i<iterations;i++)
		{
			if (i == 0)
			{
				//xekina na upologizeis me prwto pinaka paragwghs-arxikopoihshs
				play<<<no_blocks,no_threads>>>(devin, devout);
			}
			else
			{
				//sinexise na upologizeis me ton pinaka epomenhs genias
				play<<<no_blocks,no_threads>>>(devtemp, devout);
			}
			//antigrafh tou pinakas epomenhs genias ston pinaka exwdou ths gpu(eswterika ths gpu)
			hipMemcpy(devtemp, devout, size * size * sizeof(int), hipMemcpyDeviceToDevice);

			//antigrafi tou pinaka exwdou apo gpu->cpu
			hipMemcpy(output, devout, size * size * sizeof(int), hipMemcpyDeviceToHost);

			//apeikonish apotelesmatwn
			if (animate == true)
			{
				system("clear");
				print_board(output, size, i);
				usleep(100000);
			}
		}
		printf("Unoptimized run\n");
	}
	// 2h ekdosi me shared memmory,xrisimopoiei enan 3o pinaka gia tous upologismous
	else
	{
		for (int i = 0;i<iterations;i++)
		{
			if (i == 0)
			{
				//xekina na upologizeis me prwto pinaka paragwghs-arxikopoihshs
				play_with_shared_memory<<<no_blocks,no_threads,shared_board_size>>>(devin, devout, size);
			}
			else
			{
				//sinexise na upologizeis me ton pinaka epomenhs genias
				play_with_shared_memory<<<no_blocks,no_threads,shared_board_size>>>(devtemp, devout, size);
			}
			//antigrafh tou pinakas epomenhs genias ston pinaka exwdou ths gpu(eswterika ths gpu)
			hipMemcpy(devtemp, devout, size * size * sizeof(int), hipMemcpyDeviceToDevice);
			//antigrafi tou pinaka exwdou apo gpu->cpu
			hipMemcpy(output, devout, size * size * sizeof(int), hipMemcpyDeviceToHost);

			//apeikonish apotelesmatwn
			if (animate == true)
			{
				system("clear");
				print_board(output, size, i);
				usleep(100000);
			}
		}
	}

	// antigrafi tou apotelesmatos apo gpu->cpu
	hipMemcpy(output, devout, size * size * sizeof(int), hipMemcpyDeviceToHost);

	if (print)
		print_board(output, size, iterations);

	// Ypologise ton xrono ektelesis
	gettimeofday(&tv2, NULL);
	printf ("Total time in kernel = %f seconds\n",(double) (tv2.tv_usec - tv1.tv_usec) / 1000000 + (double) (tv2.tv_sec - tv1.tv_sec));



	// Free device memory
	hipFree(devin);
	hipFree(devout);
	hipFree(devtemp);


    return 0;
}

int main(int argc, char* argv[])
{
	SIZE = 0, ITERATIONS = 0, ANIMATE = -1, BLOCKS = 0, THREADS = 0, UNOPTIMIZED = 0, SEED = -1, PRINT = -1;
	arg_parse(argc, argv);
	run();
	return 0;
}
